#include "hip/hip_runtime.h"

//#include <MeqNodes/ThrustPointSourceVisibility.h>


#include <TimBase/LofarTypedefs.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sort.h>
#include <thrust/for_each.h>

#include <vector>
#include <complex>
#include <string>

// HACKHACKAHCAHCAKHCKAHCKACHKACHKACHAKCHAAAAA get rid of....
#include <cstdio>

// this is a test comment to see if my git-svn thing works!

namespace Meq {


    /***************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    struct index_flatten : public thrust::unary_function<int,int>
    {
        int inDim [4];
        int outDim [4];

        int inMults [4];
        int outMults [4];

          
        index_flatten(int aInDim, int bInDim, int cInDim, int dInDim, int aOutDim, int bOutDim, int cOutDim, int dOutDim){
            inDim[0] = aInDim;
            inDim[1] = bInDim;
            inDim[2] = cInDim;
            inDim[3] = dInDim;
            outDim[0] = aOutDim;
            outDim[1] = bOutDim;
            outDim[2] = cOutDim;
            outDim[3] = dOutDim;

            outMults[0] = outDim[0]*outDim[1]*outDim[2]*outDim[3];
            outMults[1] = outDim[1]*outDim[2]*outDim[3];
            outMults[2] = outDim[2]*outDim[3];
            outMults[3] = outDim[3];

            inMults[0] = inDim[0]*inDim[1]*inDim[2]*inDim[3];
            inMults[1] = inDim[1]*inDim[2]*inDim[3];
            inMults[2] = inDim[2]*inDim[3];
            inMults[3] = inDim[3];

        }

          
        __host__ __device__
        int operator()(int x) const
            {
                int total = 0;
                int out [4];
                out[0] = (x);
                out[0] /= outMults[1];

                total += out[0]*outMults[1];

                out[1] = (x - total);
                out[1] /= outMults[2];

                total += out[1]*outMults[2];

                out[2] = (x - total);
                out[2] /= outMults[3];

                total += out[2]*outMults[3];

                out[3] = (x - total);

                for (int i = 0 ; i < 4 ; i++) {
                    if (inDim[i] == 1)
                        out[i] = 0;
                }

                return out[0]*inMults[1] + out[1]*inMults[2] + out[2]*inMults[3] + out[3];

            }
    };


    /***************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/

    typedef thrust::tuple<double, // freq
                          double, // u
                          double, // v
                          double, // w
                          //double3, // lmn
                          double, // l
                          double, // m
                          double, // n
                          double2 // B
                          > PSVTuple;

    struct psv_functor : public thrust::unary_function<PSVTuple,double2>
    {
        double _2pi_over_c;//-casa::C::_2pi / casa::C::c;

        psv_functor(double _2pi_o_c) {
            _2pi_over_c = _2pi_o_c;
        }

        template <typename Tuple>
        __host__ __device__
        double2 operator()(Tuple t)
        {
            //double argument = _2pi_over_c*d_freq[f]*(d_u[t]*d_lmn[s*3+0]+d_v[t]*d_lmn[s*3+1]+d_w[t]*d_lmn[s*3+2]);
            //double realVal = sin(argument);
            //double imagVal = cos(argument);
            //d_output_complex[the_index+0] =  + d_B_complex[b_index*2+1]*realVal + d_B_complex[b_index*2+0]*imagVal;
            //d_output_complex[the_index+1] =  + d_B_complex[b_index*2+1]*imagVal - d_B_complex[b_index*2+0]*realVal;

            // D[i] = A[i] + B[i] * C[i];
            //thrust::get<8>(t) = thrust::get<0>(t);// + thrust::get<1>(t) * thrust::get<2>(t);

            #define FREQ thrust::get<0>(t)
            #define D_U thrust::get<1>(t)
            #define D_V thrust::get<2>(t)
            #define D_W thrust::get<3>(t)
            //#define D_LMN thrust::get<4>(t)
            //#define D_B thrust::get<5>(t)
            #define D_L thrust::get<4>(t)
            #define D_M thrust::get<5>(t)
            #define D_N thrust::get<6>(t)
            #define D_B thrust::get<7>(t)
            
            double argument = _2pi_over_c*FREQ*(D_U*D_L+D_V*D_M+D_W*D_N);
            double realVal = sin(argument);
            double imagVal = cos(argument);

            double2 ret;
            ret.x = D_B.y*realVal + D_B.x*imagVal;
            ret.y = D_B.y*imagVal + D_B.x*realVal;

            return ret;
        }
    };


    struct double2_add {

        __host__ __device__
        double2 operator()(const double2& a, const double2& b) const {

            double2 c;
            c.x = a.x + b.x;
            c.y = a.y + b.y;
            return c;
        }

    };

    /***************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    std::string runCUDAPointSourceVisibilityThrust(int nsrcs,
                                                   int nfreq,
                                                   int ntime,
                                                   thrust::device_vector<double>& d_freq,
                                                   thrust::device_vector<double>& d_u,
                                                   thrust::device_vector<double>& d_v,
                                                   thrust::device_vector<double>& d_w,
                                                   //thrust::device_vector<double3>& d_lmn,
                                                   thrust::device_vector<double>& d_l,
                                                   thrust::device_vector<double>& d_m,
                                                   thrust::device_vector<double>& d_n,
                                                   thrust::device_vector<double2>& d_b,
                                                   thrust::host_vector<double2>& h_output, 
                                                   double _2pi_over_c) {

        
      typedef thrust::device_vector<double>::iterator DoubleIterator;
      typedef thrust::device_vector<double2>::iterator DComplexIterator;
      typedef thrust::device_vector<double3>::iterator D3VectorIterator;
        
      typedef thrust::counting_iterator<int> IntCountIterator;

      IntCountIterator counter(0);

      //std::cout << "start of thrusting" << std::endl;

      // j   s   f   t |(B) 
      // 0   0   0   0 | 0
      // 0   0   0   1 | 0
      // 0   0   1   0 | 1
      // 0   0   1   1 | 1
      // 0   0   2   0 | 2
      // 0   0   2   1 | 2
      // 0   1   0   0 | 3
      // 0   1   0   1 | 3
      // 0   1   1   0 | 4
      // 0   1   1   1 | 4
      // 0   1   2   0 | 5
      // 0   1   2   1 | 5
      // 0   2   0   0 | 6
      // ...
      // 1   0   0   0 | 1*nsrc*nfreq*ntime
      // 1   0   0   1 | 1*nsrc*nfreq*ntime
      // 1   0   1   0 | 1*nsrc*nfreq*ntime + 1
      // ...      
      // 2   0   0   0 | 2*nsrc*nfreq*ntime
      // ... 
      // 1   0   1   0 | 1*nsrc*nfreq*ntime + 1
      // ...      
      // 2   0   0   0 | 2*nsrc*nfreq*ntime
      // ...      

      int unreduced_output_size = 4*nfreq*ntime*nsrcs;
      int reduced_output_size   = 4*nfreq*ntime;

      // "[]" means indexing and access like normal, "()" means the index will be flatened
      // output will be indexed on [p]([freq][time])
      // processing will operate on one of each of ([j][nsrc][freq][time])
      // B will be indexed on ([j][nsrc][freq]) - so access muct be expanded to include [time]

      typedef thrust::transform_iterator<index_flatten, IntCountIterator> TransformedIndexIterator;
      TransformedIndexIterator B_index_trans_it   (counter, index_flatten(4,1,nfreq,nsrcs,  4,ntime,nfreq,nsrcs));
      TransformedIndexIterator j_index_trans_it   (counter, index_flatten(4,1,1,1,          4,ntime,nfreq,nsrcs));
      TransformedIndexIterator time_index_trans_it(counter, index_flatten(1,ntime,1,1,      4,ntime,nfreq,nsrcs));
      TransformedIndexIterator freq_index_trans_it(counter, index_flatten(1,1,nfreq,1,      4,ntime,nfreq,nsrcs));
      TransformedIndexIterator src_index_trans_it (counter, index_flatten(1,1,1,nsrcs,      4,ntime,nfreq,nsrcs));

      TransformedIndexIterator jft_index_trans_it (counter, index_flatten(4,ntime,nfreq,1,  4,ntime,nfreq,nsrcs));

      // Need inputs to out[j]([f][t]) from unreduced_out([j][f][t][s]) from freq[f] u[t] v[t] w[t] l[s] m[s] n[s] b([j][s][f]) 

      typedef thrust::permutation_iterator<DoubleIterator, TransformedIndexIterator> PermuteTransformDoubleIterator;
      typedef thrust::permutation_iterator<DComplexIterator, TransformedIndexIterator> PermuteTransformDComplexIterator;
      typedef thrust::permutation_iterator<D3VectorIterator, TransformedIndexIterator> PermuteTransformD3VectorIterator;

      PermuteTransformDoubleIterator perm_freq_it(d_freq.begin(), freq_index_trans_it);
      PermuteTransformDoubleIterator perm_u_it   (d_u.begin(),    time_index_trans_it);
      PermuteTransformDoubleIterator perm_v_it   (d_v.begin(),    time_index_trans_it);
      PermuteTransformDoubleIterator perm_w_it   (d_w.begin(),    time_index_trans_it);
      //PermuteTransformD3VectorIterator perm_lmn_it   (d_lmn.begin(),    src_index_trans_it);
      PermuteTransformDoubleIterator perm_l_it   (d_l.begin(),    src_index_trans_it);
      PermuteTransformDoubleIterator perm_m_it   (d_m.begin(),    src_index_trans_it);
      PermuteTransformDoubleIterator perm_n_it   (d_n.begin(),    src_index_trans_it);
      PermuteTransformDComplexIterator perm_B_it (d_b.begin(),    B_index_trans_it);


      typedef thrust::tuple<PermuteTransformDoubleIterator,   // freq
                            PermuteTransformDoubleIterator,   // u
                            PermuteTransformDoubleIterator,   // v
                            PermuteTransformDoubleIterator,   // w
                            //PermuteTransformD3VectorIterator,   // lmn
                            PermuteTransformDoubleIterator,   // l
                            PermuteTransformDoubleIterator,   // m
                            PermuteTransformDoubleIterator,   // n
                            PermuteTransformDComplexIterator//, // B 
                            //DoubleIterator                    // unreduced_output
                            > // result
          PSVIteratorTuple;


      typedef thrust::zip_iterator<PSVIteratorTuple> PSVZipIterator;

      PSVZipIterator data_zip_it (thrust::make_tuple(
                                      perm_freq_it,
                                      perm_u_it,
                                      perm_v_it,
                                      perm_w_it,
                                      //perm_lmn_it,
                                      perm_l_it,
                                      perm_m_it,
                                      perm_n_it,
                                      perm_B_it//,
                                      //d_unreduced_output.begin()
                                      ));
      
      PSVZipIterator data_zip_it_end (thrust::make_tuple(
                                          perm_freq_it+unreduced_output_size,
                                          perm_u_it+unreduced_output_size,
                                          perm_v_it+unreduced_output_size,
                                          perm_w_it+unreduced_output_size,
                                          //perm_lmn_it+unreduced_output_size,
                                          perm_l_it+unreduced_output_size,
                                          perm_m_it+unreduced_output_size,
                                          perm_n_it+unreduced_output_size,
                                          perm_B_it+unreduced_output_size//,
                                          //d_unreduced_output.end()
                                          ));

      std::cout << "created iterators" << std::endl;

      thrust::device_vector<double2> d_unreduced_output(unreduced_output_size);

      std::cout << "device temp unreduced output alloced ("<< unreduced_output_size << "*" <<sizeof(double2) << ") = "<< (unreduced_output_size*sizeof(double2)) << " bytes" << std::endl;

      thrust::transform (data_zip_it, data_zip_it_end, d_unreduced_output.begin(), psv_functor(_2pi_over_c));

      std::cout << "calc transformed" << std::endl;
      //thrust::transform_iterator<psv_functor ,PSVZipIterator> psv_transform_it(data_zip_it, psv_functor(_2pi_over_c));

      thrust::device_vector<double2> d_output(reduced_output_size);
      thrust::device_vector<int> d_output_keys(reduced_output_size);
      std::cout << "device output + keys alloced" << std::endl;


      size_t avail;
      size_t total;
      hipMemGetInfo( &avail, &total );
      size_t used = total - avail;

      std::cout << "Device memory total: " << total << std::endl;
      std::cout << "Device memory avail: " << avail << std::endl;
      std::cout << "Device memory used:  " << used << std::endl;

      size_t will_use = (unreduced_output_size*sizeof(double2))+(unreduced_output_size*3*sizeof(unsigned int));

      std::cout << "reduction will temporarily alloc at least ("<< unreduced_output_size << "*" <<sizeof(double2) << " + " << unreduced_output_size << "*3*" << sizeof(unsigned int) << ") = "<< will_use << " bytes" << std::endl;

      if (will_use > avail) {

          char a [256];
          sprintf(a,"Not enough memory for reduce by key (%u bytes available, %u bytes needed)", avail, will_use);
          return std::string(a);

      }

      thrust::equal_to<int> equal_pred;
      thrust::reduce_by_key(jft_index_trans_it,
                            jft_index_trans_it + unreduced_output_size,
                            d_unreduced_output.begin(),
                            //thrust::make_transform_iterator(data_zip_it,  
                            //                                psv_functor(_2pi_over_c)
                            //    ),
                            //psv_transform_it,
                            d_output_keys.begin(),
                            d_output.begin(),
                            equal_pred,
                            double2_add()
                            );
      std::cout << "key reduced with reduce_by_key" << std::endl;

      h_output = d_output;



        //------------start old----------------

      // thrust::host_vector<double2> h_unreduced_output(d_unreduced_output);

      // for (int i = 0 ; i < reduced_output_size ; i++) {
      //     h_output[i].x = 0;
      //     h_output[i].y = 0;
      //     for (int s = 0 ; s < nsrcs ; s++){
      //         h_output[i].x += h_unreduced_output[(i*nsrcs)+s].x;
      //         h_output[i].y += h_unreduced_output[(i*nsrcs)+s].y;
      //     }
      //     }

        //------------end old------------------
      

      // NOTE: using make_transform_iterator(data_zip_it, psv_functor(_2pi_over_c)) should be more efficient since it will transform in place rather than doing a transform run and store, then a read and reduce. Owing to the number of parameters, the formal parameter space overflows, tried to condense l, m & n into a double3 vector instead of 3 double vectors but same problem arrises (I thought the double3 would use one pointer space - instead of 3 - but it seems it passes the double3 directly). Due to the double3 vector not collesing as well as 3 double vectors, it goes slightly slower (0.13 sec/tile vs. 0.12 sec/tile type of difference). So I'm just returning to using 3 double vectors and 2 passes.

      return "";
      
    }

}
