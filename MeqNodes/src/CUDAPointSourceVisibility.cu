#include "hip/hip_runtime.h"

//#include <MeqNodes/CUDAPointSourceVisibility.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <vector>
#include <complex>
#include <string>
#include <TimBase/LofarTypedefs.h>

#include <MeqNodes/CUDAPointSourceVisibilityCommon.h>

// HACKHACKAHCAHCAKHCKAHCKACHKACHKACHAKCHAAAAA get rid of....
#include <cstdio>

// this is a test comment to see if my git-svn thing works!

texture<float, 1, hipReadModeElementType> texRef;

namespace Meq {


    /****************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/

    // inline __device__ int CUDAgetMultiDimIndex(int a, int aT, int b, int bT, int c, int cT, int d, int dT, int e, int eT) {
    //     return a*bT*cT*dT*eT + b*cT*dT*eT + c*dT*eT + d*eT + e;
    // }

    __device__ __host__ __inline__ int getMultiDimIndex(int a, int aT, int b, int bT, int c, int cT, int d, int dT, int e, int eT) {
        return a*bT*cT*dT*eT + b*cT*dT*eT + c*dT*eT + d*eT + e;
    }

    __device__ __host__ __inline__ int getMultiDimIndex(int a, int aT, int b, int bT, int c, int cT, int d, int dT) {
        return a*bT*cT*dT + b*cT*dT + c*dT + d;
    }

    __device__ __host__ __inline__ int getMultiDimIndex(int a, int aT, int b, int bT, int c, int cT) {
        return a*bT*cT + b*cT + c;
    }

    __device__ __host__ __inline__ int getMultiDimIndex(int a, int aT, int b, int bT) {
        return a*bT + b;
    }

    /****************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    __device__ __host__ int get_B_index(int s, int nsrcs, 
                                        int f, int nfreq, 
                                        int j, int num_matrix_elements){
    
        // return getMultiDimIndex( s,               nsrcs,             
        //                          f,               nfreq,
        //                          j,               num_matrix_elements);

        return getMultiDimIndex( j, num_matrix_elements,
                                 s, nsrcs,             
                                 f, nfreq);
    }

    /****************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    __device__ __host__ int get_sf_jones_index(int s, int nsrcs, 
                                               int f, int nfreq){

        return getMultiDimIndex(s, nsrcs, 
                                f, nfreq);

    }


    /****************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    __device__ __host__ __inline__ int get_intermediate_output_index(int s, int nsrcs, 
                                                                     int t, int ntime, 
                                                                     int f, int nfreq, 
                                                                     int j, int num_matrix_elements){


        // return getMultiDimIndex(s, nsrcs, 
        //                         t, ntime,
        //                         f, nfreq,
        //                         j, num_matrix_elements);

        // return getMultiDimIndex(j, num_matrix_elements,
        //                         t, ntime,
        //                         f, nfreq,
        //                         s, nsrcs );


        // return getMultiDimIndex(j, num_matrix_elements,
        //                         s, nsrcs, 
        //                         t, ntime,
        //                         f, nfreq);

        // return getMultiDimIndex(s, nsrcs,
        //                         j, num_matrix_elements,
        //                         f, nfreq,
        //                         t, ntime);

        return getMultiDimIndex(j, num_matrix_elements,
                                s, nsrcs, 
                                f, nfreq,
                                t, ntime);



    }

    /****************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    __device__ __host__ __inline__ int get_shared_mem_index(int t, int ntime, 
                                                            int f, int nfreq, 
                                                            int j, int num_matrix_elements){


        // return getMultiDimIndex(t, ntime,
        //                         f, nfreq,
        //                         j, num_matrix_elements);


        // return getMultiDimIndex(j, num_matrix_elements,
        //                         t, ntime,
        //                         f, nfreq);

        return getMultiDimIndex(j, num_matrix_elements,
                                f, nfreq,
                                t, ntime);

    }
    /****************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    __device__ __host__ __inline__ int get_output_index(int t, int ntime, 
                                                        int f, int nfreq, 
                                                        int j, int num_matrix_elements){


        return getMultiDimIndex(t, ntime,
                                f, nfreq,
                                j, num_matrix_elements);


    }


    /****************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/

    inline __device__ __host__ dim3 fromNormalToAdjustedDim(const dim3& normal) {
        return dim3(normal.x, normal.y*normal.z, 1);
    }

    /****************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/

    inline __device__ __host__ dim3 fromAdjustedToNormalDim(const dim3& adjusted, const dim3& originalGrid) {
        return dim3(adjusted.x, adjusted.y%originalGrid.y, adjusted.y/originalGrid.y);
    }



    //  /***************************************************************************
    //  **
    //  ** Author: Richard Baxter
    //  **
    //  ****************************************************************************/
    // __global__ void CUDAPointSourceVisibilityKernel_K(dim3 desiredGridDim, 
    //                                                   lmn_t* d_lmn, 
    //                                                   double2* d_B_complex,
    //                                                   int nsrcs, 
    //                                                   int srcs_offset, 
    //                                                   int srcs_per_thread,
    //                                                   double* d_uvw, 
    //                                                   double* d_time, 
    //                                                   int ntime, 
    //                                                   double* d_freq, 
    //                                                   int nfreq, 
    //                                                   int num_matrix_elements, 
    //                                                   double2* d_intermediate_output_complex, 
    //                                                   double _2pi_over_c
    //     ) {   


    //     double* d_u = d_uvw;
    //     double* d_v = d_uvw+ntime;
    //     double* d_w = d_uvw+(ntime*2);


    //     dim3 actualBlockIdx = fromAdjustedToNormalDim(blockIdx, desiredGridDim);
    //     #define ADJ_X actualBlockIdx.x
    //     #define ADJ_Y actualBlockIdx.y
    //     #define ADJ_Z actualBlockIdx.z


    //     int s_i = ((ADJ_X*blockDim.x) + threadIdx.x);
    //     int s = s_i+srcs_offset;
    //     int t = ((ADJ_Y*blockDim.y) + threadIdx.y);
    //     int f = ((ADJ_Z*blockDim.z) + threadIdx.z);

    //     if (t < ntime && f < nfreq) 
    //     {
    //         if (s < nsrcs)
    //         {

    //             double argument = _2pi_over_c*(d_u[t]*d_lmn[s].x+d_v[t]*d_lmn[s].y+d_w[t]*d_lmn[s].z);

    //             double realVal;
    //             double imagVal;
    //             sincos(d_freq[f]*argument, &realVal, &imagVal);

    //             for( int j=0; j<num_matrix_elements; ++j ){


    //                 // calcuating B*exp(...) = B*E = (B.r+jB.i)(E.r+jE.i) = (B.r*E.r - B.i*E.i) + j(B.r*E.i + B.i*E.r)
    //                 //                                                    = (B.i*E.r + B.r*E.i) - j(B.i*E.i + B.r*E.r)

    //                 int b_index = get_B_index(s, nsrcs, 
    //                                           f, nfreq,
    //                                           j, num_matrix_elements);

    //                 int the_index = get_intermediate_output_index(s_i, nsrcs, // must address this via the index
    //                                                               t,   ntime,
    //                                                               f,   nfreq,
    //                                                               j,   num_matrix_elements);

    //                 d_intermediate_output_complex[the_index].x = 
    //                     (+ d_B_complex[b_index].y*realVal + d_B_complex[b_index].x*imagVal);
    //                 d_intermediate_output_complex[the_index].y = 
    //                     (+ d_B_complex[b_index].y*imagVal - d_B_complex[b_index].x*realVal);
            


    //             }
    //         }
    //     }

        
    // }



    //  /***************************************************************************
    //  **
    //  ** Author: Richard Baxter
    //  **
    //  ****************************************************************************/
    // __global__ void CUDAPointSourceVisibilityKernel_Smear(dim3 desiredGridDim, 
    //                                                       lmn_t* d_lmn, 
    //                                                       double2* d_B_complex,
    //                                                       int nsrcs, 
    //                                                       int srcs_offset, 
    //                                                       int srcs_per_thread,
    //                                                       double* d_uvw, 
    //                                                       double* d_duvw, 
    //                                                       double* d_time, 
    //                                                       int ntime, 
    //                                                       double* d_freq, 
    //                                                       int nfreq, 
    //                                                       double* d_df_over_2, 
    //                                                       double* d_f_dt_over_2, 
    //                                                       int num_matrix_elements, 
    //                                                       double2* d_intermediate_output_complex, 
    //                                                       double _2pi_over_c
    //     ) {   


    //     double* d_u = d_uvw;
    //     double* d_v = d_uvw+ntime;
    //     double* d_w = d_uvw+(ntime*2);

    //     double* d_du = d_duvw;
    //     double* d_dv = d_duvw+ntime;
    //     double* d_dw = d_duvw+(ntime*2);

    //     dim3 actualBlockIdx = fromAdjustedToNormalDim(blockIdx, desiredGridDim);
    //     #define ADJ_X actualBlockIdx.x
    //     #define ADJ_Y actualBlockIdx.y
    //     #define ADJ_Z actualBlockIdx.z




    //     int s_i = ((ADJ_X*blockDim.x) + threadIdx.x);
    //     int s = s_i+srcs_offset;
    //     int t = ((ADJ_Y*blockDim.y) + threadIdx.y);
    //     int f = ((ADJ_Z*blockDim.z) + threadIdx.z);



    //     if (t < ntime && f < nfreq) 
    //     {


    //         if (s < nsrcs)
    //         {


    //             //double dargument = _2pi_over_c*(d_du[t]*d_lmn[s].x+d_dv[t]*d_lmn[s].y+d_dw[t]*d_lmn[s].z);
    //             //double argument = _2pi_over_c*(d_u[t]*d_lmn[s].x+d_v[t]*d_lmn[s].y+d_w[t]*d_lmn[s].z);
    //             double E_jones = 1.0;


    //             double dphi = d_f_dt_over_2[t] * _2pi_over_c*(d_du[t]*d_lmn[s].x+d_dv[t]*d_lmn[s].y+d_dw[t]*d_lmn[s].z);
    //             if (dphi != 0.0) 
    //                 E_jones = sin(dphi)/dphi;

    //             double dpsi = d_df_over_2[f] * _2pi_over_c*(d_u[t]*d_lmn[s].x+d_v[t]*d_lmn[s].y+d_w[t]*d_lmn[s].z);
    //             if (dpsi != 0.0)
    //                 E_jones *= sin(dpsi)/dpsi;



    //             for( int j=0; j<num_matrix_elements; ++j ){


    //                 int the_index = get_intermediate_output_index(s_i, nsrcs, // must address this via the index
    //                                                               t,   ntime,
    //                                                               f,   nfreq,
    //                                                               j,   num_matrix_elements);

    //                 d_intermediate_output_complex[the_index].x *= E_jones;
    //                 d_intermediate_output_complex[the_index].y *= E_jones;
        



    //             }
    //         }
    //     }

        
    // }

     /***************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    __global__ void 
    //__launch_bounds__(256, 3)
        CUDAPointSourceVisibilityKernel(dim3 desiredGridDim, 
                                        lmn_t* d_lmn, 
                                        double2* d_B_complex,
                                        int nsrcs, 
                                        int nslots, 
                                        int srcs_offset, 
                                        int srcs_per_thread,
                                        double* d_uvw, 
                                        double* d_duvw, 
                                        double* d_time, 
                                        int ntime, 
                                        double* d_freq, 
                                        int nfreq, 
                                        double* d_df_over_2, 
                                        double* d_f_dt_over_2, 
                                        double* d_e_jones,
                                        double* d_e_jones_h,
                                        int num_matrix_elements, 
                                        double2* d_intermediate_output_complex, 
                                        double _2pi_over_c
        ) {   


        double* d_u = d_uvw;
        double* d_v = d_uvw+ntime;
        double* d_w = d_uvw+(ntime*2);
// #define D_U(t) d_u[t]
// #define D_V(t) d_v[t]
// #define D_W(t) d_w[t]
// #define D_U(t) d_uvw[t]
// #define D_V(t) d_uvw[ntime + t]
// #define D_W(t) d_uvw[(ntime*2)+t]

        double* d_du = d_duvw;
        double* d_dv = d_duvw+ntime;
        double* d_dw = d_duvw+(ntime*2);
// #define D_DU(t) d_du[t]
// #define D_DV(t) d_dv[t]
// #define D_DW(t) d_dw[t]
// #define D_DU(t) (d_duvw[t])
// #define D_DV(t) (d_duvw[ntime + t])
// #define D_DW(t) (d_duvw[(ntime*2)+t])


        //double* d_f_dt_over_2 = 0;
    
        // Axes:
        //   x: sources
        //   y: time
        //   z: freq


        //#define ADJ_X (blockIdx.x)
        //#define ADJ_Y (blockIdx.y%desiredGridDim.y)
        //#define ADJ_Z (blockIdx.y/desiredGridDim.y)

        dim3 actualBlockIdx = fromAdjustedToNormalDim(blockIdx, desiredGridDim);
        #define ADJ_X actualBlockIdx.x
        #define ADJ_Y actualBlockIdx.y
        #define ADJ_Z actualBlockIdx.z



        int s_i = ((ADJ_X*blockDim.x) + threadIdx.x); // input index
//#ifdef MULTI_SRC_PER_THREAD
//        int s_start_i = ((ADJ_X*blockDim.x) + threadIdx.x);
//#endif
#ifndef MULTI_SRC_PER_THREAD
        int s = s_i+srcs_offset;
#endif
        int t = ((ADJ_Y*blockDim.y) + threadIdx.y); // = t_i since calcs per thread = 1
        int f = ((ADJ_Z*blockDim.z) + threadIdx.z); // = f_i since calcs per thread = 1
        //int t_i = t;
        //int f_i = f;

#ifdef SHARED_MEMORY
        #define NTIME_SHARED blockDim.y
        #define NFREQ_SHARED blockDim.z
        int t_si = threadIdx.y; // si = shared (memory) output index
        int f_si = threadIdx.z;
#endif

        //printf("(%i,%i,%i) %i\n", s_i, t, f);
        //for (int s = srcsID_start ; s < srcsID_start+1 && s < nsrcs ; ++s) {
            
        //for (int t = timeID_start ; t < timeID_start+1 && t < ntime; ++t)  {

        //for (int f = freqID_start ; f < freqID_start+1 && f < nfreq ; ++f) {

        // int s = srcsID_start;
        // int t = timeID_start;
        // int f = freqID_start;

        //printf("INFO (%i, %i, %i) \t(t %i/%i, f %i/%i)\n", s_i, t, f, t_si, NTIME_SHARED, f_si, NFREQ_SHARED);

        if (t < ntime && f < nfreq) 
        {

#ifdef SHARED_MEMORY

            //printf("(%i, %i, %i) SHARED!!!!!\n", s_i, t_si, f_si);
            extern __shared__ double2 shared_mem []; // size blockDim.y*blockDim.z*num_matrix_elements
            //printf("(%i, %i, %i) REALLY SHARED!!!!!\n", s_i, t_si, f_si);

//#ifdef MULTI_SRC_PER_THREAD
            for( int j=0; j<num_matrix_elements; ++j ){
                int share_index = get_shared_mem_index(t_si, NTIME_SHARED,
                                                       f_si, NFREQ_SHARED,
                                                       j, num_matrix_elements);

                //printf("(%i, %i, %i) \tshare index: %i \t(t %i/%i, f %i/%i, j %i/%i)\n", s_i, t, f, share_index, t_si, NTIME_SHARED, f_si, NFREQ_SHARED, j, num_matrix_elements);
                shared_mem[share_index].x = 0;
                shared_mem[share_index].y = 0;
            }
//#endif
#endif


#ifdef MULTI_SRC_PER_THREAD
//#pragma unroll 64
                for (int s = s_i*srcs_per_thread; s < (s_i+1)*srcs_per_thread && s < nsrcs ; ++s) 
#endif
#ifndef MULTI_SRC_PER_THREAD
                if (s < nsrcs)
#endif

            {

                double argument = _2pi_over_c*(d_u[t]*d_lmn[s].x+d_v[t]*d_lmn[s].y+d_w[t]*(d_lmn[s].z-1));


                double smearFactor = 1.0;

                if (d_duvw) {
//#define DARGUMENT (_2pi_over_c*(D_DU(t)*d_lmn[s].x+D_DV(t)*d_lmn[s].y+D_DW(t)*d_lmn[s].z));

                    double dargument = _2pi_over_c*(d_du[t]*d_lmn[s].x+d_dv[t]*d_lmn[s].y+d_dw[t]*(d_lmn[s].z-1));
                    double dphi = d_f_dt_over_2[t] * dargument;
                    if (dphi != 0.0) 
                        smearFactor = sin(dphi)/dphi;

                    double dpsi = d_df_over_2[f] * argument;
                    if (dpsi != 0.0)
                       smearFactor *= sin(dpsi)/dpsi;

                }

                // double argument = _2pi_over_c*d_freq[f]*(d_u[t]*d_lmn[s].x+d_v[t]*d_lmn[s].y+d_w[t]*d_lmn[s].z);
                // double realVal = sin(argument);
                // double imagVal = cos(argument);

                double realVal; 
                double imagVal;
                sincos(d_freq[f]*argument, &realVal, &imagVal);

                double e_jones_term = 1.0;
                double e_jones_term_H = 1.0;

                if (d_e_jones) {

                    int e_j_index = get_sf_jones_index(s,   nsrcs,
                                                       f,   nfreq);
                    e_jones_term   = d_e_jones   [e_j_index];
                    e_jones_term_H = d_e_jones_h [e_j_index];
                }

                for( int j=0; j<num_matrix_elements; ++j ){

                    // TODO use sincos

                    // calcuating B*exp(...) = B*E = (B.r+jB.i)(E.r+jE.i) = (B.r*E.r - B.i*E.i) + j(B.r*E.i + B.i*E.r)
                    //                                                    = (B.i*E.r + B.r*E.i) - j(B.i*E.i + B.r*E.r)

                    int b_index = get_B_index(s, nsrcs, 
                                              f, nfreq,
                                              j, num_matrix_elements);

#define REAL_TERM e_jones_term*(+ d_B_complex[b_index].y*realVal + d_B_complex[b_index].x*imagVal)*smearFactor*e_jones_term_H
#define IMAG_TERM e_jones_term*(+ d_B_complex[b_index].y*imagVal - d_B_complex[b_index].x*realVal)*smearFactor*e_jones_term_H

// #define REAL_TERM (( + d_B_complex[b_index].y*realVal + d_B_complex[b_index].x*imagVal)*smearFactor)
// #define IMAG_TERM (( + d_B_complex[b_index].y*imagVal - d_B_complex[b_index].x*realVal)*smearFactor)
// #define REAL_TERM -1
// #define IMAG_TERM -1


#ifndef SHARED_MEMORY
                    int the_index = get_intermediate_output_index(s_i, nslots, // must address this via the index
                                                                  t,   ntime,
                                                                  f,   nfreq,
                                                                  j,   num_matrix_elements);

  #ifdef MULTI_SRC_PER_THREAD
                    d_intermediate_output_complex[the_index].x += REAL_TERM;
                    d_intermediate_output_complex[the_index].y += IMAG_TERM;
  #endif

  #ifndef MULTI_SRC_PER_THREAD
                    d_intermediate_output_complex[the_index].x = REAL_TERM;
                    d_intermediate_output_complex[the_index].y = IMAG_TERM;
  #endif
                    
#endif

#ifdef SHARED_MEMORY

                    int share_index = get_shared_mem_index(t_si, NTIME_SHARED,
                                                           f_si, NFREQ_SHARED,
                                                           j,    num_matrix_elements);

                    //printf("(%i, %i, %i) \tshare index: %i \t(t %i, f %i, j %i)\n", s_i, t, f, share_index, t_si, f_si, j);

  #ifdef MULTI_SRC_PER_THREAD
                    shared_mem[share_index].x += REAL_TERM;
                    shared_mem[share_index].y += IMAG_TERM;
  #endif

  #ifndef MULTI_SRC_PER_THREAD
                    shared_mem[share_index].x = REAL_TERM;
                    shared_mem[share_index].y = IMAG_TERM;
  #endif
                    //shared_mem[share_index].x = ADJ_Y;
                    //shared_mem[share_index].y = ntime;  
#endif

#undef REAL_TERM
#undef IMAG_TERM


                }
            }
#ifdef SHARED_MEMORY
            for( int j=0; j<num_matrix_elements; ++j ){
                int the_index = get_intermediate_output_index(s_i, nslots, 
                                                              t,   ntime,
                                                              f,   nfreq,
                                                              j,   num_matrix_elements);
                int share_index = get_shared_mem_index(t_si, NTIME_SHARED,
                                                       f_si, NFREQ_SHARED,
                                                       j,    num_matrix_elements);
                d_intermediate_output_complex[the_index] = shared_mem[share_index];
            }
#endif
        }

        //}
        //}
        
    }


     /***************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    __global__ void CUDAPointSourceVisibilityReductionKernel(dim3 desiredGridDim, 
                                                             int nsrcs, 
                                                             int nslots, 
                                                             int srcs_offset, 
                                                             int srcs_per_thread,
                                                             int ntime, 
                                                             int nfreq, 
                                                             int num_matrix_elements, 
                                                             double2* d_intermediate_output_complex,
                                                             int reducers,
                                                             int gap) {   
        
        // Axes:
        //   x: sources (reductor)
        //   y: time
        //   z: freq

        dim3 actualBlockIdx = fromAdjustedToNormalDim(blockIdx, desiredGridDim);


        int xThreadIdx = ((actualBlockIdx.x*blockDim.x) + threadIdx.x);
        int yThreadIdx = ((actualBlockIdx.y*blockDim.y) + threadIdx.y);
        int zThreadIdx = ((actualBlockIdx.z*blockDim.z) + threadIdx.z);

        int s_i = xThreadIdx*gap*2;
        int s_i2 = s_i+gap;
        int t = yThreadIdx;
        int f = zThreadIdx;

                                
        //for (int s = srcsID_start ; s < srcsID_start+1 && s < nsrcs ; ++s) {
            
        //for (int t = timeID_start ; t < timeID_start+1 && t < ntime; ++t)  {

        //for (int f = freqID_start ; f < freqID_start+1 && f < nfreq ; ++f) {

        // int s = srcsID_start;
        // int s2 = srcsID2_start;
        // int t = timeID_start;
        // int f = freqID_start;
        if ((s_i*srcs_per_thread)+srcs_offset < nsrcs && (s_i2*srcs_per_thread)+srcs_offset < nsrcs && s_i != s_i2 && t < ntime && f < nfreq) { //TODO - only need n-1 threads if n is odd

            for( int j=0; j<num_matrix_elements; ++j ){

                int the_index = get_intermediate_output_index(s_i, nslots, 
                                                              t,   ntime,
                                                              f,   nfreq,
                                                              j,   num_matrix_elements);

                int the_index2 = get_intermediate_output_index(s_i2, nslots, 
                                                               t,    ntime,
                                                               f,    nfreq,
                                                               j,    num_matrix_elements);

                d_intermediate_output_complex[the_index].x += d_intermediate_output_complex[the_index2].x;
                d_intermediate_output_complex[the_index].y += d_intermediate_output_complex[the_index2].y;


            }
        }
        //}
        //}
        //}

        


    }

     /***************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    __global__ void CUDAAddToOutputKernel(dim3 desiredGridDim, 
                                          int nsrcs, 
                                          int nslots, 
                                          int ntime, 
                                          int nfreq, 
                                          int num_matrix_elements, 
                                          double2* d_intermediate_output_complex, 
                                          double2* d_output_complex) {   
        
        // Axes:
        //   x: sources (reductor = 0)
        //   y: time
        //   z: freq

        dim3 actualBlockIdx = fromAdjustedToNormalDim(blockIdx, desiredGridDim);


        int xThreadIdx = ((actualBlockIdx.x*blockDim.x) + threadIdx.x);
        int yThreadIdx = ((actualBlockIdx.y*blockDim.y) + threadIdx.y);
        int zThreadIdx = ((actualBlockIdx.z*blockDim.z) + threadIdx.z);

        int s_i = xThreadIdx;
        int t = yThreadIdx;
        int f = zThreadIdx;
                 
        //printf("(%i, %i, %i)\n", s_i,t,f);      

        if (s_i == 0 && t < ntime && f < nfreq) {

            for( int j=0; j<num_matrix_elements; ++j ){

                int the_i_index = get_intermediate_output_index(s_i, nslots, 
                                                                t,   ntime,
                                                                f,   nfreq,
                                                                j,   num_matrix_elements);

                int the_index = get_output_index(t,  ntime,
                                                 f,  nfreq,
                                                 j,  num_matrix_elements);

                //printf("%i \t<-> %i\n");

                //d_intermediate_output_complex[the_i_index].x = 3;
                d_output_complex[the_index].x = d_intermediate_output_complex[the_i_index].x;
                d_output_complex[the_index].y = d_intermediate_output_complex[the_i_index].y;

                //printf("(%i, %i, %i:%i) \tinput: %i \toutput:%i  \t threadIdx.y: %i  \t blockIdx.y %i \t actualBlockIdx.y %i\n", s_i,t,f,j, the_i_index, the_index, threadIdx.y, blockIdx.y, actualBlockIdx.y);

            }
        }

        //d_output_complex[t].x = t;
        //d_output_complex[t].y = -2;
    }


     /***************************************************************************
     **
     ** Author: Richard Baxter
     **
     ****************************************************************************/
    std::string runCUDAPointSourceVisibilityKernel(lmn_t* d_lmn, 
                                                   double2* d_B_complex, 
                                                   int nsrcs,
                                                   int nslots,
                                                   int nsrcs_per_slot,
                                                   int nslots_per_run,
                                                   double* d_uvw,
                                                   double* d_duvw, 
                                                   double* d_time, 
                                                   int ntime, 
                                                   double* d_freq,  
                                                   int nfreq, 
                                                   double* d_df_over_2, 
                                                   double* d_f_dt_over_2,
                                                   double* d_e_jones,
                                                   double* d_e_jones_h,
                                                   double2* d_intermediate_output_complex,
                                                   double2* d_output_complex, 
                                                   int NUM_MATRIX_ELEMENTS,
                                                   double _2pi_over_c,/*make this a constant variable rather than a parameter*/ 
                                                   std::complex<double> ** pout) {


        //cdebug(0) << "Executing Kernel" << endl;

        //double* h_smear_parameters[5] = {d_du, d_dv, d_dw, d_df_over_2, d_f_dt_over_2};
        //hipMemcpyToSymbol(HIP_SYMBOL(d_smear_parameters), h_smear_parameters, 5*sizeof(double*), 0, hipMemcpyHostToDevice);

           
        printf("address of d_duvw: %i\n", d_duvw);

        hipError_t error;
        error = hipGetLastError  ();
        if (error != 0) {
            return std::string("CUDA constant memory copy error: ") + std::string(hipGetErrorString (error));
        }

        int time_threads = 8;
        int freq_threads = 32;//64;
#ifdef SHARED_MEMORY
        freq_threads = 16; // HACK - on 1.x devices 64 threads means there won't be enough shared memory
#endif

        int remaining_slots = nslots;

        int src_offset = 0;

        while(remaining_slots != 0) {

            //=======================================================================================
            int nslots_this_run = nslots_per_run;
            if (remaining_slots < nslots_this_run)
                nslots_this_run = remaining_slots;

            remaining_slots -= nslots_this_run;
            
            printf("running %i slots at %i srcs per slot\n", nslots_this_run, nsrcs_per_slot);

            // Axes are:               srcs,            time,         freq

            int calcsPerProblem [3] = {nslots_this_run,  ntime,        nfreq};
            //int calcsPerGrid    [3] = {128,   1024,  64};

            int calcsPerBlock   [3] = {1,                time_threads, freq_threads}; // product of these will be the number of threads
            int calcsPerThread  [3] = {1,                1,     1}; // MUST = 1. so (1, 1, 1)

            printf("Number of calculation per grid: s) %i, t) %i, f) %i\n", calcsPerProblem[0], calcsPerProblem[1], calcsPerProblem[2]);

#ifdef SHARED_MEMORY
            //hipFuncSetCacheConfig(reinterpret_cast<const void*>(CUDAPointSourceVisibilityKernel), hipFuncCachePreferShared);

            size_t shared_memory = time_threads * freq_threads * NUM_MATRIX_ELEMENTS * sizeof(double2);
            printf("Requesting %i bytes of shared memory\n", shared_memory);
#endif


            for (int i = 0 ; i < 3 ;++i) {
                //if (calcsPerProblem[i] < calcsPerBlock[i]) return "Elements in \'Calculations per Grid\' must be less than or equal to the corresponding element in \'Calculations per Problem\'";
                //FailWhen (calcsPerGrid[i] < calcsPerBlock[i], "Elements in \'Calculations per Block\' must be less than or equal to the corresponding element in \'Calculations per Grid\'");
                if(calcsPerBlock[i] < calcsPerThread[i])
                    return "Elements in \'Calculations per Thread\' must be less than or equal to the corresponding element in \'Calculations per Block\'";
            }

            int gridSize  [3];
            int blockSize [3];
            for (int i = 0 ; i < 3 ;++i) {
                gridSize[i]  = ( (calcsPerProblem[i]-1) / calcsPerBlock[i]  ) + 1;
                blockSize[i] = (  (calcsPerBlock[i]-1)   / calcsPerThread[i] ) + 1;
            }

            //cdebug(0) << "calcsPerProblem: " << calcsPerProblem[0] << ", " << calcsPerProblem[1] << ", " << calcsPerProblem[2] << endl; 
            //cdebug(0) << "calcsPerGrid:    " << calcsPerGrid[0] << ", " << calcsPerGrid[1] << ", " << calcsPerGrid[2] << endl;
            //cdebug(0) << "calcsPerBlock:   " << calcsPerBlock[0] << ", " << calcsPerBlock[1] << ", " << calcsPerBlock[2] << endl;
            //cdebug(0) << "calcsPerThread:  " << calcsPerThread[0] << ", " << calcsPerThread[1] << ", " << calcsPerThread[2] << endl;


            dim3 gridDim (gridSize[0],  gridSize[1],  gridSize[2]);
            dim3 blockDim(blockSize[0], blockSize[1], blockSize[2]);
            dim3 threadDim(gridSize[0]*blockSize[0], gridSize[1]*blockSize[1], gridSize[2]*blockSize[2]);
            dim3 calcsPerThreadDim(calcsPerThread[0], calcsPerThread[1], calcsPerThread[2]);
            //int srcsDim = gridDim.x*blockDim.x*calcsPerThreadDim.x;
            //int timeDim = gridDim.y*blockDim.y*calcsPerThreadDim.y;
            //int freqDim = gridDim.z*blockDim.z*calcsPerThreadDim.z;      

            //cdebug(0) << "Running Kernel gridDim(" << gridDim.x << ", " << gridDim.y << ", " << gridDim.z << ") blockDim(" << blockDim.x << ", " << blockDim.y << ", " << blockDim.z << ")" << ") threadDim(" << threadDim.x << ", " << threadDim.y << ", " << threadDim.z << ")" << endl;

            dim3 adjGridDim = fromNormalToAdjustedDim(gridDim); 
            //nothingKernel<<<gridDim, blockDim>>>();
            //hipDeviceSynchronize();
            printf("adjgrid  %ix%ix%i\n", adjGridDim.x, adjGridDim.y, adjGridDim.z);
            printf("grid     %ix%ix%i\n", gridDim.x, gridDim.y, gridDim.z);
            printf("block    %ix%ix%i = %i threads\n", blockDim.x, blockDim.y, blockDim.z, blockDim.x* blockDim.y* blockDim.z);

            CUDAPointSourceVisibilityKernel<<<adjGridDim, blockDim
#ifdef SHARED_MEMORY
                , shared_memory
#endif
                                           >>> (gridDim, 
                                                d_lmn, 
                                                d_B_complex, 
                                                nsrcs, 
                                                nslots_this_run, 
                                                src_offset, 
                                                nsrcs_per_slot, 
                                                d_uvw,
                                                d_duvw,
                                                d_time, ntime, 
                                                d_freq, nfreq,
                                                d_df_over_2, d_f_dt_over_2,
                                                d_e_jones, d_e_jones_h,
                                                NUM_MATRIX_ELEMENTS, 
                                                d_intermediate_output_complex, 
                                                _2pi_over_c
                                                );
            hipDeviceSynchronize();

            // CUDAPointSourceVisibilityKernel_K<<<adjGridDim, blockDim
            //                                  >>> (gridDim, 
            //                                       d_lmn, 
            //                                       d_B_complex, 
            //                                       nsrcs, 
            //                                       src_offset, 
            //                                       nsrcs_per_slot, 
            //                                       d_uvw,
            //                                       d_time, ntime, 
            //                                       d_freq, nfreq,
            //                                       NUM_MATRIX_ELEMENTS, 
            //                                       d_intermediate_output_complex, 
            //                                       _2pi_over_c
            //                                      );


            // CUDAPointSourceVisibilityKernel_Smear<<<adjGridDim, blockDim
            //                                      >>> (gridDim, 
            //                                           d_lmn, 
            //                                           d_B_complex, 
            //                                           nsrcs, 
            //                                           nslots_this_run, 
            //                                           src_offset, 
            //                                           nsrcs_per_slot, 
            //                                           d_uvw,
            //                                           d_duvw,
            //                                           d_time, ntime, 
            //                                           d_freq, nfreq,
            //                                           d_df_over_2, d_f_dt_over_2,
            //                                           NUM_MATRIX_ELEMENTS, 
            //                                           d_intermediate_output_complex, 
            //                                           _2pi_over_c
            //                                          );
            error = hipGetLastError  ();
            if (error != 0) {
                return std::string("CUDA runtime error: ") + std::string(hipGetErrorString (error));
            }

            //=======================================================================================

            printf("Reduction\n");
            int level = 0;
            int reductors = nslots_this_run;
            while(reductors != 1) {

                if (reductors%2 == 1)
                    reductors ++;
                reductors>>=1;

                int gap = 1<<level;
                level++;

                printf("lvl: %i\n", level);
                printf("red: %i\n", reductors);
                printf("gap: %i\n", gap);
                printf("\n");

                
                gridDim.x = reductors;
                adjGridDim = fromNormalToAdjustedDim(gridDim); 
                
                printf("adjgrid  %ix%ix%i\n", adjGridDim.x, adjGridDim.y, adjGridDim.z);
                printf("grid     %ix%ix%i\n", gridDim.x, gridDim.y, gridDim.z);
                printf("block    %ix%ix%i = %i threads\n", blockDim.x, blockDim.y, blockDim.z, blockDim.x* blockDim.y* blockDim.z);
                CUDAPointSourceVisibilityReductionKernel<<<adjGridDim, blockDim>>> (gridDim, nsrcs, nslots_this_run, src_offset, nsrcs_per_slot,  ntime, nfreq, NUM_MATRIX_ELEMENTS, d_intermediate_output_complex, reductors, gap);
                hipDeviceSynchronize();

                hipError_t error;
                error = hipGetLastError  ();
                if (error != 0) 
                    return std::string("CUDA runtime error after reduction step: ") + std::string(hipGetErrorString (error));


            } //while(reductors != 1) {
            
            //=======================================================================================
            gridDim.x = 1;
            adjGridDim = fromNormalToAdjustedDim(gridDim); 

            printf("Adding to output array on device\n");
            printf("adjgrid  %ix%ix%i\n", adjGridDim.x, adjGridDim.y, adjGridDim.z);
            printf("grid     %ix%ix%i\n", gridDim.x, gridDim.y, gridDim.z);
            printf("block    %ix%ix%i = %i threads\n", blockDim.x, blockDim.y, blockDim.z, blockDim.x* blockDim.y* blockDim.z);


            CUDAAddToOutputKernel<<<adjGridDim, blockDim>>> (gridDim, nsrcs, nslots_this_run, ntime, nfreq, NUM_MATRIX_ELEMENTS, d_intermediate_output_complex, d_output_complex);
            hipDeviceSynchronize();

            error = hipGetLastError  ();
            if (error != 0) 
                return std::string("CUDA runtime error after output copy: ") + std::string(hipGetErrorString (error));
            
            //-----------------------------------------------------------------------------------

            src_offset += nslots_this_run*nsrcs_per_slot;
        } // end while(remaining_srcs != 0)

        //printf("Copying from device to host\n");
         std::vector<double2> output_complex(ntime*nfreq*NUM_MATRIX_ELEMENTS);
        if (hipMemcpy(&(output_complex[0]), d_output_complex, sizeof(double2)*ntime*nfreq*NUM_MATRIX_ELEMENTS , hipMemcpyDeviceToHost) != hipSuccess) {
            return "Memcopy error copying data from device (d_output_complex -> output_complex) : " + std::string(hipGetErrorString (hipGetLastError  ()));
        }
        printf("Copying to pout %i\n", ntime);
        for (int t = 0 ; t < ntime ; ++t) {
            for (int f = 0 ; f < nfreq ; ++f) {

                for( int j=0; j<NUM_MATRIX_ELEMENTS; ++j ){

                    int the_index = get_output_index(t,  ntime,
                                                     f,  nfreq,
                                                     j,  NUM_MATRIX_ELEMENTS);
                             
                    pout[j][t*nfreq + f] = 
                        std::complex<double>(
                            output_complex[the_index].x, 
                            output_complex[the_index].y
                            );
                    //cdebug(0) << "new total pout[" << j << "]([" << t << "][" << f << "]) " << pout[j][t*freqDim + f] << endl;
                }
            }
        }
        printf("Done\n");
        //========================================================================================


        return "";

    }


}
